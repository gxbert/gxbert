#include "GXBenchmarker.hh"
#include "GXBenchmarker_gpu.hh"

#include "VecHepDefs.hh"
#include "GXTrackHandler.hh"
#include "GXTrack.hh"
#include "GXCurand.hh"

namespace gxbert {

void GXBenchmarker::RunCuda()
{
  int nDevice;
  bool cudaEnabled = false;

  hipGetDeviceCount(&nDevice);
  if(nDevice > 0) {
    hipDeviceReset();
    cudaEnabled = true;
  }
  else {
    printf("Waning: No Cuda Capable Device ...");
  }

  //cuda event timing
  hipEvent_t start, stop;
  hipEventCreate (&start);
  hipEventCreate (&stop);

  GXTrack* itrack_aos = (GXTrack*) malloc(fNtracks*sizeof(GXTrack));
  GXTrack* otrack_aos = (GXTrack*) malloc(fNtracks*sizeof(GXTrack));

  //allocate memory for input/output tracks
  GXTrack *itrack_d;
  GXTrack *otrack_d;

  hipMalloc((void**)&itrack_d, fNtracks*sizeof(GXTrack));
  hipMalloc((void**)&otrack_d, fNtracks*sizeof(GXTrack));

  //set the default number of threads and thread blocks - should be setable
  int theNBlocks  =  26;
  int theNThreads = 192;

  //prepare random engines on the device
  Random_t* randomStates = 0;
  hipMalloc(&randomStates, theNBlocks*theNThreads* sizeof(hiprandState));
  GXCurand_Init(randomStates, time(NULL), theNBlocks, theNThreads);

  float elapsedTotal[kNumberTest];
  float elapsedEventTime[kNumberTest];
  double resultTotal[kNumberTest];

  double *result_d;

  double *result_h = (double*) calloc(theNBlocks*theNThreads, sizeof(double));
  // malloc(fNtracks*sizeof(double));
  hipMalloc((void**)&result_d,theNBlocks*theNThreads*sizeof(double));

  for (int k = 0; k < kNumberTest; ++k) {
    elapsedTotal[k] = 0.;
    resultTotal[k] = 0.;
  }

  fTrackHandler->SetRandomStream(1);
  fTrackHandler->GenerateRandomTracks(fNtracks,fMinP, fMaxP);
  GXTrack* track_aos = fTrackHandler->GetAoSTracks();

  for (unsigned r = 0; r < fRepetitions; ++r) {

    for (int k = 0; k < kNumberTest; ++k) {

      fTrackHandler->CopyAoSTracks(track_aos,itrack_aos,fNtracks);
      hipMemcpy(itrack_d, track_aos, fNtracks*sizeof(GXTrack), hipMemcpyHostToDevice);
      
      elapsedEventTime[k] = 0.0;
      
      if(cudaEnabled) {
        hipEventRecord (start,0);

        //call CUDA kernels
        CudaKernelFunc[k](theNBlocks, theNThreads, randomStates,
                          fNtracks, itrack_d, otrack_d, result_d);

        hipEventRecord (stop,0);
        hipEventSynchronize (stop);
        hipEventElapsedTime (&elapsedEventTime[k],start,stop);

        //copy the result for varification
        hipMemcpy(result_h,result_d,theNBlocks*theNThreads*sizeof(double),hipMemcpyDeviceToHost);
        for(int i = 0 ; i < theNBlocks*theNThreads ; ++i) resultTotal[k] += result_h[i];

      }
      elapsedTotal[k] += elapsedEventTime[k]; //ms
      
      hipMemcpy(itrack_aos, itrack_d, fNtracks*sizeof(GXTrack), hipMemcpyDeviceToHost);
      hipMemcpy(otrack_aos, otrack_d, fNtracks*sizeof(GXTrack), hipMemcpyDeviceToHost);
    }
  }

  for (int k = 0; k < kNumberTest; ++k) {
    printf("%s  Cuda   Total time of %3d reps = %7.4f msec result = %6.3f\n", 
      TestName[k], fRepetitions, elapsedTotal[k], resultTotal[k]);
  }

  //clean up: destory cuda event and free memory on device and host
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(randomStates);
  hipFree(itrack_d);
  hipFree(otrack_d);
  hipFree(result_d);
  free(result_h);

  free(itrack_aos);
  free(otrack_aos);
}

} // end of gxbert namespace
