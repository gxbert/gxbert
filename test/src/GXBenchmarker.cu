#include "GXBenchmarker.h"
#include "GXBenchmarker_gpu.h"

#include "VecHepDefs.h"
#include "GXTrackHandler.h"
#include "GXTrack.h"
#include "GXCurand.h"

namespace gxbert {

void GXBenchmarker::RunCuda()
{
  int nDevice;
  bool cudaEnabled = false;

  hipGetDeviceCount(&nDevice);
  if(nDevice > 0) {
    hipDeviceReset();
    cudaEnabled = true;
  }
  else {
    printf("Waning: No Cuda Capable Device ...");
  }

  //cuda event timing
  hipEvent_t start, stop;
  hipEventCreate (&start);
  hipEventCreate (&stop);

  GXTrack* itrack_aos = (GXTrack*) malloc(fNtracks*sizeof(GXTrack));
  GXTrack* otrack_aos = (GXTrack*) malloc(fNtracks*sizeof(GXTrack));

  //allocate memory for input/output tracks
  GXTrack *itrack_d;
  GXTrack *otrack_d;

  hipMalloc((void**)&itrack_d, fNtracks*sizeof(GXTrack));
  hipMalloc((void**)&otrack_d, fNtracks*sizeof(GXTrack));

  //set the default number of threads and thread blocks - should be setable
  int theNBlocks  =  26;
  int theNThreads = 192;

  //prepare random engines on the device
  Random_t* randomStates = 0;
  hipMalloc(&randomStates, theNBlocks*theNThreads* sizeof(hiprandState));
  GXCurand_Init(randomStates, time(NULL), theNBlocks, theNThreads);

  float elapsedTotal[kNumberTest];
  float elapsedEventTime[kNumberTest];

  for (int k = 0; k < kNumberTest; ++k) elapsedTotal[k] = 0.;

  for (unsigned r = 0; r < fRepetitions; ++r) {

    fTrackHandler->GenerateRandomTracks(fNtracks,fMinP, fMaxP);

    GXTrack* track_aos = fTrackHandler->GetAoSTracks();
    fTrackHandler->SortAoSTracksByEnergy(track_aos,fNtracks);

    for (int k = 0; k < kNumberTest; ++k) {

      fTrackHandler->CopyAoSTracks(track_aos,itrack_aos,fNtracks);
      hipMemcpy(itrack_d, track_aos, fNtracks*sizeof(GXTrack), hipMemcpyHostToDevice);
      
      elapsedEventTime[k] = 0.0;
      
      if(cudaEnabled) {
        hipEventRecord (start,0);

        //call CUDA kernels
        CudaKernelFunc[k](theNBlocks, theNThreads, randomStates,
                          fNtracks, itrack_d, otrack_d);

        hipEventRecord (stop,0);
        hipEventSynchronize (stop);
        hipEventElapsedTime (&elapsedEventTime[k],start,stop);
      }
      elapsedTotal[k] += elapsedEventTime[k]/1000.; //ms to second
      
      hipMemcpy(itrack_aos, itrack_d, fNtracks*sizeof(GXTrack), hipMemcpyDeviceToHost);
      hipMemcpy(otrack_aos, otrack_d, fNtracks*sizeof(GXTrack), hipMemcpyDeviceToHost);
    }
  }

  for (int k = 0; k < kNumberTest; ++k) {
    printf("%s  Cuda Total time of %3d reps = %7.4f sec\n", 
      TestName[k], fRepetitions, elapsedTotal[k]);
  }

  //clean up: destory cuda event and free memory on device and host
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(randomStates);
  hipFree(itrack_d);
  hipFree(otrack_d);

  free(itrack_aos);
  free(otrack_aos);
}

} // end of gxbert namespace
