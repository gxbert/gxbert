#include "hip/hip_runtime.h"
#include "GXCurand.h"

#include <iostream>

namespace gxbert {
inline namespace cuda {

__global__
void GXCurand_Init_Kernel(Random_t *randomStates, unsigned long seed) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(seed, tid, 0, &randomStates[tid]);
}

} // end of cuda namespace

bool GXCurand_Init(Random_t *randomStates, unsigned long seed, 
                   int blocksPerGrid, int threadsPerBlock) 
{
  int kstatus = 0;

  gxbert::cuda::GXCurand_Init_Kernel<<<blocksPerGrid,threadsPerBlock>>>
                                                     (randomStates,seed);

  hipError_t err = hipGetLastError();
  if ( hipSuccess != err ) {
    fprintf(stderr,"GXCurand_Init cudaCheckError() failed at %s : %i : %s\n",
	    __FILE__, __LINE__, hipGetErrorString(err));
    exit(-1);
  }

  kstatus = hipDeviceSynchronize();
  if (kstatus) std::cout << "GXCurand_Init status = " << kstatus << "\n";
  
  return true;
}

} // end of gxbert namespace
