#include "hip/hip_runtime.h"
#include "GXThreeVector.h"
#include "LorentzVector.h"
#include "GXTrack.h"

#include "VecRng/RngDefs.h"

namespace gxbert {
inline namespace cuda {

__global__
void KernelTest01(Random_t* devStates, 
                  int nTrackSize, GXTrack* itrack, GXTrack* otrack, double *result)
{
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int sid = tid;

  GXThreeVector<double> vec3a;
  GXThreeVector<double> vec3b;

 __shared__ double sum[26*192];
  double tmp = 0;

  while (tid < nTrackSize) {
    vec3a.Set(itrack[tid].x,itrack[tid].y,itrack[tid].z);
    vec3b.Set(itrack[tid].px,itrack[tid].py,itrack[tid].pz);
    tmp += vec3a.Dot(vec3b);
    tid += blockDim.x * gridDim.x;
  }

  sum[sid] = tmp;

  __syncthreads();

  //do reduction on CPU
  result[sid] = sum[sid];
}

__global__
void KernelTest02(Random_t* devStates, 
                  int nTrackSize, GXTrack* itrack, GXTrack* otrack, double *result)
{
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int sid = tid;

  GXThreeVector<double> vec3a;
  GXThreeVector<double> vec3b;
  GXThreeVector<double> vec3c;

  __shared__ double sum[26*192];
  double tmp = 0;

  while (tid < nTrackSize) {
    vec3a.Set(itrack[tid].x,itrack[tid].y,itrack[tid].z);
    vec3b.Set(itrack[tid].px,itrack[tid].py,itrack[tid].pz);
    vec3c = vec3b.RotateUz(vec3a.Unit());
    tmp += vec3c.Perp2();
    tid += blockDim.x * gridDim.x;
  }

  sum[sid] = tmp;

  __syncthreads();

  //do reduction on CPU
  result[sid] = sum[sid];

}

__global__
void KernelBoost(Random_t* devStates, 
                  int nTrackSize, GXTrack* itrack, GXTrack* otrack, double *result)
{
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int sid = tid;

  LorentzVector<double> vec4a;
  LorentzVector<double> vec4b;
  GXThreeVector<double> vec3;

  __shared__ double sum[26*192];
  double tmp = 0;

  while (tid < nTrackSize) {
    vec4a.Set(itrack[tid].px,itrack[tid].py,itrack[tid].pz,
              itrack[tid].E + itrack[tid].m);
    vec4b = vec4a.Boost(vec4a.BoostVector());
    tmp +=  vec4b.Perp2();
    tid += blockDim.x * gridDim.x;
  }

  sum[sid] = tmp;

  __syncthreads();

  //do reduction on CPU
  result[sid] = sum[sid];

}

} // end namespace cuda

// Cuda wrapper

void CudaTest01(int blocksPerGrid, int threadsPerBlock, Random_t* devStates,
 		int nTrackSize, GXTrack* itrack, GXTrack* otrack, double *result) 
{
  gxbert::cuda::KernelTest01<<<blocksPerGrid, threadsPerBlock>>>(devStates,
                nTrackSize,itrack,otrack, result);
}

void CudaTest02(int blocksPerGrid, int threadsPerBlock, Random_t* devStates,
 		int nTrackSize, GXTrack* itrack,GXTrack* otrack, double *result) 
{
  gxbert::cuda::KernelTest02<<<blocksPerGrid, threadsPerBlock>>>(devStates,
                nTrackSize,itrack,otrack,result);
}

void CudaBoost(int blocksPerGrid, int threadsPerBlock, Random_t* devStates,
 		int nTrackSize, GXTrack* itrack,GXTrack* otrack, double *result) 
{
  gxbert::cuda::KernelBoost<<<blocksPerGrid, threadsPerBlock>>>(devStates,
                nTrackSize,itrack,otrack,result);
}

} // end namespace gxbert
