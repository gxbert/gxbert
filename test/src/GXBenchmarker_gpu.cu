#include "hip/hip_runtime.h"
#include "GXThreeVector.h"
#include "VecHepDefs.h"
#include "GXTrack.h"

namespace gxbert {
inline namespace cuda {

__global__
void KernelTest01(Random_t* devStates, 
                  int nTrackSize, GXTrack* itrack, GXTrack* otrack)
{
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  GXThreeVector<double> vec3a;
  GXThreeVector<double> vec3b;

  double result = 0;

  while (tid < nTrackSize) {
    vec3a.Set(itrack[tid].x,itrack[tid].y,itrack[tid].z);
    result += vec3a.Dot(vec3b);
    tid += blockDim.x * gridDim.x;
  }
}

__global__
void KernelTest02(Random_t* devStates, 
                  int nTrackSize, GXTrack* itrack, GXTrack* otrack)
{
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  GXThreeVector<double> vec3a;
  GXThreeVector<double> vec3b(1.,0.,0.);
  GXThreeVector<double> vec3c;

  while (tid < nTrackSize) {
    vec3a.Set(itrack[tid].x,itrack[tid].y,itrack[tid].z);
    vec3c = vec3a.RotateUz(vec3b);
    tid += blockDim.x * gridDim.x;
  }
}

} // end namespace cuda

// Cuda wrapper

void CudaTest01(int blocksPerGrid, int threadsPerBlock, Random_t* devStates,
 		int nTrackSize, GXTrack* itrack, GXTrack* otrack) 
{
  gxbert::cuda::KernelTest01<<<blocksPerGrid, threadsPerBlock>>>(devStates,
                nTrackSize,itrack,otrack);
}

void CudaTest02(int blocksPerGrid, int threadsPerBlock, Random_t* devStates,
 		int nTrackSize, GXTrack* itrack,GXTrack* otrack) 
{
  gxbert::cuda::KernelTest02<<<blocksPerGrid, threadsPerBlock>>>(devStates,
                nTrackSize,itrack,otrack);
}

} // end namespace gxbert
